#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdlib.h>
#include <stdio.h>

#define TPB 32
#define M 100

__device__
float distance(float x1, float x2){
	return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	//printf("blockIdx:%2d,blockDim:%2d,threadIdx:%2d,i = %2d:dist from %f to %f.\n",
	//	blockIdx.x,blockDim.x,threadIdx.x, i, ref, x, d_out[i]);
}

void distanceArray(float *out, float *in, float ref, int len){
	hipEvent_t startCpy, stopCpy;
	hipEvent_t startKernel, stopKernel;
	hipEventCreate(&startCpy);
	hipEventCreate(&stopCpy);
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	float *d_in = 0;
	float *d_out = 0;

	hipMalloc(&d_in, len*sizeof(float));
	hipMalloc(&d_out, len*sizeof(float));
	
	hipEventRecord(startCpy);
	for(int i=0;i < M;++i)
		hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(stopCpy);
	
	hipEventRecord(startKernel);
	distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
	hipEventRecord(stopKernel);

	hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stopCpy);
	hipEventSynchronize(stopKernel);
	
	float copyTime = 0;
	hipEventElapsedTime(&copyTime,startCpy,stopCpy);
	float kernelTime = 0;
	hipEventElapsedTime(&kernelTime,startKernel,stopKernel);
	printf("copy time:%f (ms)\nkernel time:%f (ms)\n",copyTime,kernelTime);

	hipFree(d_in);
	hipFree(d_out);
}
