#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#define TPB 32
#define M 100

__device__
float distance(float x1, float x2){
	return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	//printf("blockIdx:%2d,blockDim:%2d,threadIdx:%2d,i = %2d:dist from %f to %f.\n",
	//	blockIdx.x,blockDim.x,threadIdx.x, i, ref, x, d_out[i]);
}

void distanceArray(float *out, float *in, float ref, int len){
	float *d_in = 0;
	float *d_out = 0;

	hipMalloc(&d_in, len*sizeof(float));
	hipMalloc(&d_out, len*sizeof(float));
	
	clock_t copyBegin = clock();
	for(int i=0;i < M;++i)
		hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
	clock_t copyEnd = clock();
	
	clock_t kernelBegin = clock();
	distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
	clock_t kernelEnd = clock();

	hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);
	
	double copyTime = ((double)(copyEnd - copyBegin)) / CLOCKS_PER_SEC;
	double kernelTime = ((double)(kernelEnd - kernelBegin)) / CLOCKS_PER_SEC;
	printf("copy time:%f (ms)\nkernel time:%f (ms)\n",copyTime*1000,kernelTime*1000);

	hipFree(d_in);
	hipFree(d_out);
}
