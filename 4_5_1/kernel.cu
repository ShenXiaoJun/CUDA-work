#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32

__device__
unsigned char clip(int n){ return n > 255 ? 255 : (n < 0 ? 0 : n);}

__global__
void distanceKernel(uchar4 *d_out, int w, int h, float AtoBsalce, int2 Bpos){
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	if((c>=w) || (r>=h)) return;
	const int i = c + r*w;
	const int Bdist = sqrtf((c - Bpos.x)*(c - Bpos.x) + (r - Bpos.y)*(r - Bpos.y));
	const unsigned char intensity = clip(255 - Bdist);

	d_out[i].x = intensity*AtoBsalce;
	d_out[i].y = 0;
	d_out[i].z = intensity*(1-AtoBsalce);
	d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, int2 Apos, int2 Bpos){
	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3((w + TX -1)/TX, (h + TY - 1)/TY);
	const int AtoBdist = sqrtf((Apos.x - Bpos.x)*(Apos.x - Bpos.x) + (Apos.y - Bpos.y)*(Apos.y - Bpos.y));
	float AtoBscaleTmp = AtoBdist/sqrtf(w*w+h*h);
	float AtoBsalce = AtoBscaleTmp > 1 ? 1 : AtoBscaleTmp;
	distanceKernel<<<gridSize, blockSize>>>(d_out, w, h, AtoBsalce, Bpos);
}
