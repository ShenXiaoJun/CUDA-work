
#include <hip/hip_runtime.h>
#include <stdio.h>
#define W 32
#define H 32
#define D 32
#define TX 8
#define TY 8
#define TZ 8

int divUp(int a, int b){ return (a + b - 1) / b; }

__device__
float distance(int c, int r, int s, float3 pos){
	return sqrtf((c - pos.x)*(c - pos.x) + (r - pos.y)*(r - pos.y) +
			(s - pos.z)*(s - pos.z));
}

__global__
void distanceKernel(float *d_out, int w,int h, int d,float3 pos){
	const int c = blockIdx.x*blockDim.x + threadIdx.x;// column
	const int r = blockIdx.y*blockDim.y + threadIdx.y;// row
	const int s = blockIdx.z*blockDim.z + threadIdx.z;// stack
	const int i = c + r*w + s*w*h;
	if ((c >= w)||(r >= h) || (s >= d)) return;

	d_out[i] = distance(c, r, s, pos);
	printf("d_out[%d]:%f\n",i,d_out[i]);
}

int main(){
	float *out = (float *)calloc(W*H*D, sizeof(float));
	float *d_out=0;
	hipMalloc(&d_out, W*H*D*sizeof(float));
	
	const float3 pos = {0.0f, 0.0f, 0.0f};
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(W, TX), divUp(H, TY), divUp(D, TZ));

	distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, D, pos);
	
	hipMemcpy(out, d_out, W*H*D*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_out);
	free(out);
	return 0;
}
