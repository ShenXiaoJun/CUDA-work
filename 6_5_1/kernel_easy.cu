#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 12
#define ATOMIC 1

__global__
void even_sort(int *d_b, const int n){
	int tid = threadIdx.x;//线程从0开始编号
	if(1 == (tid + 1 ) % 2)//第奇数个轮回
	{
		if(d_b[tid] > d_b[tid + 1] && tid + 1 < n){
			int tmp = d_b[tid];
			d_b[tid] = d_b[tid + 1];
			d_b[tid + 1] = tmp;
		}
	}
	__syncthreads();
}

__global__
void odd_sort(int *d_b, const int n){
	int tid = threadIdx.x;//线程从0开始编号
	if(0 == (tid + 1 ) % 2)//第奇数个轮回
	{
		if(d_b[tid] > d_b[tid + 1] && tid + 1 < n){
			int tmp = d_b[tid];
			d_b[tid] = d_b[tid + 1];
			d_b[tid + 1] = tmp;
		}
	}
}

void dotLauncher(int *b, int n){
	int *d_b = NULL;

	hipMalloc(&d_b, n*sizeof(int));

	hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);
	for(int i = 0; i < n; ++i){
		even_sort<<<1, n, 0>>>(d_b, n);
		odd_sort<<<1, n, 0>>>(d_b, n);
	}
	hipMemcpy(b, d_b, n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_b);
}
