#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 12
// 未完成
__global__
void sort_even(int *d_b, int n, int *d_sort_even_run){
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx >= n) return;
	const int s_idx = threadIdx.x;

	__shared__ int s_prod[TPB];
	if(abs(d_b[idx]) < abs(d_b[idx+1])){
		int tmp = d_b[idx];
		d_b[idx] = d_b[idx+1];
		d_b[idx+1] = tmp;
		s_prod[s_idx] = 1;
	} else s_prod[s_idx] = 0;
	__syncthreads();
	//每个线程块的线程0做管理
	if(s_idx == 0){
		int blockSum = 0;
		for(int j = 0; j < blockDim.x && j < (n - blockDim.x * blockIdx.x); j+=2){
			blockSum += s_prod[j];
			printf("%s,j:%d\n",__func__,j);
		}
		atomicAdd(d_sort_even_run, blockSum);
		printf("%s,Block_%d, blockSum = %d, d_sort_even_run = %d\n", __func__, blockIdx.x, blockSum, *d_sort_even_run);
	}
}

__global__
void sort_odd(int *d_b, int n, int *d_sort_odd_run){
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx + 1 >= n) return;
	const int s_idx = threadIdx.x;

	__shared__ int s_prod[TPB];
	if(abs(d_b[idx]) < abs(d_b[idx+1])){
		int tmp = d_b[idx];
		printf("idx=%d,d_b=%d,d_b+1=%d,start\n",idx,d_b[idx],d_b[idx+1]);
		d_b[idx] = d_b[idx+1];
		d_b[idx+1] = tmp;
		s_prod[s_idx] = 1;
		printf("idx=%d,d_b=%d,d_b+1=%d,end\n",idx,d_b[idx],d_b[idx+1]);
	} else s_prod[s_idx] = 0;
	__syncthreads();
	//每个线程块的线程0做管理
	if(s_idx == 0){
		int blockSum = 0;
		for(int j = 1; j < blockDim.x && j+1 < (n - blockDim.x * blockIdx.x); j+=2){
			blockSum += s_prod[j];
			if(0)printf("%s,j:%d\n",__func__,j);
		}
		atomicAdd(d_sort_odd_run, blockSum);
		printf("%s,Block_%d, blockSum = %d, d_sort_odd_run = %d\n", __func__, blockIdx.x, blockSum, *d_sort_odd_run);
	}
}

void dotLauncher(int *b, int n){
	int *d_b = NULL, *d_sort_odd_run = NULL, *d_sort_even_run = NULL;
	int sort_odd_run = 0, sort_even_run = 0;

	hipMalloc(&d_b, n*sizeof(int));
	hipMalloc(&d_sort_odd_run, sizeof(int));
	hipMalloc(&d_sort_even_run, sizeof(int));

	hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);

	//for(int i=0; i < 2*n; i++){
	for(int i=0; 1; i++){
		hipMemset(d_sort_even_run,0,sizeof(int));
		hipMemset(d_sort_odd_run,0,sizeof(int));
		sort_even<<<(n + TPB - 1)/TPB, TPB>>>(d_b, n, d_sort_even_run);
		sort_odd<<<(n + TPB - 1)/TPB, TPB>>>(d_b, n, d_sort_odd_run);
		hipMemcpy(&sort_even_run, d_sort_even_run, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&sort_odd_run, d_sort_odd_run, sizeof(int), hipMemcpyDeviceToHost);
		printf("sort_odd_run=%d, sort_even_run=%d\n",sort_odd_run ,sort_even_run);
		if(sort_odd_run + sort_even_run == 0)
			break;
	}
	hipMemcpy(b, d_b, n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_b);
	hipFree(d_sort_even_run);
	hipFree(d_sort_odd_run);
}
