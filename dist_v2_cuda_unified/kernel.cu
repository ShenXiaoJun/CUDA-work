
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
//#define N 128
//#define N 1024
//#define N 63
//#define N 65
//#define N 4096
#define TPB 32
//#define TPB 1


float scale(int i, int n){
	return ((float) i)/(n - 1);
}

__device__ 
float distance(float x1, float x2){
	return sqrt((x2-x1)*(x2-x1));
}

__global__ 
void distanceKernel(float *d_out, float *d_in, float ref){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	float x = 0;
	//if(i>N-1)
	//	return;
	x = d_in[i];
	d_out[i] = distance(x, ref);
	if(0)
	{
		printf("blockIdx:%2d,blockDim:%2d,threadIdx:%2d, i = %2d: dist from %f to %f.\n",
			blockIdx.x,blockDim.x,threadIdx.x,i, ref, x, d_out[i]);
	}
	//if(i==4095) printf("find 4095\n");
}

int main(){
	const float ref = 0.5f;
	float *in = 0;
	float *out = 0;
	
	hipMallocManaged(&in, N*sizeof(float));
	hipMallocManaged(&out, N*sizeof(float));
	
	for(int i=0;i<N;++i)
		in[i]=scale(i,N);

	distanceKernel<<<(N+TPB-1)/TPB, TPB>>>(out, in, ref);
	hipDeviceSynchronize();
	
	hipFree(in);
	hipFree(out);
	return 0;
}
