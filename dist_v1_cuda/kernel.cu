
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
#define TPB 32

__device__ float scale(int i, int n){
	return ((float) i)/(n - 1);
}

__device__ float distance(float x1, float x2){
	return sqrt((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float *d_out, float ref, int len){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = scale(i, len);
	d_out[i] = distance(x, ref);
	printf("blockIdx:%2d,blockDim:%2d,threadIdx:%2d,i = %2d: dist from %f to %f.\n",
		blockIdx.x,blockDim.x,threadIdx.x,i, ref, x, d_out[i]);
}

int main(){
	const float ref = 0.5f;
	float *d_out = 0;
	
	hipMalloc(&d_out, N*sizeof(float));

	distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);
	
	hipFree(d_out);
	return 0;
}
